#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;

//Sobel���ӱ�Ե���˺���
__global__ void sobelInCuda(unsigned char* dataIn, unsigned char* dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;

    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}

//Sobel���ӱ�Ե���CPU����
void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar* dataUp = srcImg.ptr<uchar>(i - 1);
        uchar* data = srcImg.ptr<uchar>(i);
        uchar* dataDown = srcImg.ptr<uchar>(i + 1);
        uchar* out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}

int main()
{
    Mat grayImg = imread("D:/project/image_segment_with_cuda/test.jpg", 0);

    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;

    Mat gaussImg;
    //��˹�˲�
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    double time1 = static_cast<double>(getTickCount());
    //Sobel����CPUʵ��
    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    sobel(gaussImg, dst, imgHeight, imgWidth);
    //��ʱ������
    time1 = ((double)getTickCount() - time1) / getTickFrequency();
    //�������ʱ��
    cout << "The Run Time is :" << time1<< "s" << endl;


    //CUDAʵ�ֺ�Ĵ��ص�ͼ��
    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    //����GPU�ڴ�
    unsigned char* d_in;
    unsigned char* d_out;

    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));

    //����˹�˲����ͼ���CPU����GPU
    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //��ʱ����ʼ
    double time0 = static_cast<double>(getTickCount()); 
    //���ú˺���
    sobelInCuda << <blockspergrid, threadsperblock>>> (d_in, d_out, imgheight, imgwidth);
    //sobelInCuda << <1,512 >> > (d_in, d_out, imgHeight, imgWidth);
    //��ʱ������
    time0 = ((double)getTickCount() - time0) / getTickFrequency(); 
    //�������ʱ��
    cout << "The Run Time is :" << time0 << "s" << endl; 


    //��ͼ�񴫻�GPU
    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //�ͷ�GPU�ڴ�
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}